#include "hip/hip_runtime.h"
#include <iostream>

#include "../include/warpdrive.cuh"
#include "../tools/binary_io.h"

int main(int argc, char const *argv[]) {

    using namespace warpdrive;
    using namespace std;

    if (argc < 9)
    {
        cerr << "ERROR: Not enough parameters (read \"PARAMS\" section inside the main function)" << endl;
        return -1;
    }

    //the global index type to use
    using index_t = uint64_t;

    //PARAMS
    //the size of the thread groups (must be available at compile time)
    static constexpr index_t group_size = 4;
    //output verbosity (must be available at compile time)
    static constexpr index_t verbosity = 2;
    //filename for test data (dumped with binary_io.h)
    const string  filename = argv[1];
    //length of test data
    const index_t len_data = atoll(argv[2]);
    //load factor of the hash table
    const float   load     = atof(argv[3]);
    //capacity of the hash table
    const index_t capacity = len_data/load;
    //max chaotic probing attempts
    const index_t lvl1_max = atoi(argv[4]);
    //max linear probing attempts
    const index_t lvl2_max = atoi(argv[5]);
    //number of CUDA blocks per grid
    const index_t blocks_per_grid = (atoi(argv[6]) != 0) ? atoi(argv[6]): (1UL << 31)-1;
    //number of threads per CUDA block (must be multiple of group_size)
    const index_t threads_per_block = atoi(argv[7]);
    //id of selected CUDA device
    const index_t device_id = atoi(argv[8]);

    if (verbosity > 0)
    {
        cout << "================= WARPDRIVE ================" << endl;
        cout << "================== PARAMS =================="
             << "\n(static) group_size=" << group_size
             << "\n(static) verbosity=" << verbosity
             << "\nfilename=" << filename
             << "\nlen_data=" << len_data
             << "\nload=" << load
             << "\ncapacity=" << capacity
             << "\nlvl1_max=" << lvl1_max
             << "\nlvl2_max=" << lvl2_max
             << "\nblocks_per_grid=" << blocks_per_grid
             << "\nthreads_per_block=" << threads_per_block
             << "\ndevice_id=" << device_id
             << endl;
    }

    //DECLS
    //data policy
    using data_p    = warpdrive::policies::PackedPairDataPolicy<>;
    //failure policy
    using failure_p = std::conditional<(verbosity > 1), //if verbosity < 2 ignore failures
                                       warpdrive::policies::PrintIdFailurePolicy,
                                       warpdrive::policies::IgnoreFailurePolicy>::type;

   //data types
   using data_t  = data_p::data_t;
   using key_t   = data_p::key_t;
   using value_t = data_p::value_t;

    //plan (the meat)
    using plan_t = plans::BasicPlan<group_size, //size of parallel probing group
                                    data_p,
                                    failure_p,
                                    index_t>; //index type to use>

    //config struct (probing lengths and kernel launch config)
    plan_t::config_t config(lvl1_max,
                            lvl2_max,
                            blocks_per_grid,
                            threads_per_block);

    //set the selected CUDA device
    hipSetDevice(device_id); CUERR

    //load random keys
    key_t * keys_h = new key_t[len_data];
    load_binary<key_t>(keys_h, len_data, filename);

    //the hash table
    data_t * hash_table_d; hipMalloc(&hash_table_d, sizeof(data_t)*capacity); CUERR

    //test data
    data_t * data_h = new data_t[len_data];
    data_t * data_d; hipMalloc(&data_d, sizeof(data_t)*len_data); CUERR

    //TESTS/BENCHMARKS
    if (verbosity > 0)
    {
        cout << "============= TESTS/BENCHMARK =============" << endl;
    }

    //init failure handler
    failure_p failure_handler = failure_p();
    failure_handler.init();

    //the first task to execute
    using elem_op_1 = data_p::update_op;
    static constexpr auto table_op_1 = plan_t::table_op_t::insert;

    //the second task to execute
    using elem_op_2 = data_p::nop_op;
    static constexpr auto table_op_2 = plan_t::table_op_t::retrieve;

    //init hash table
    memset_kernel
    <<<SDIV(capacity, 1024), 1024>>>
    (hash_table_d, capacity, data_t(data_p::empty_key, elem_op_1::identity));

    //init input data
    #pragma omp parallel for
    for (index_t i = 0; i < len_data; i++)
    {
        data_h[i] = data_t(keys_h[i], i+1);
    }

    hipMemcpy(data_d, data_h, sizeof(data_t)*len_data, H2D); CUERR

    //execute task
    TIMERSTART(op1)
    plan_t::table_operation<table_op_1,
                            elem_op_1>
    (data_d, len_data, hash_table_d, capacity, failure_handler, 0, config);
    TIMERSTOP(op1)

    //re-init data
    #pragma omp parallel for
    for (index_t i = 0; i < len_data; i++)
    {
        data_h[i].set_value(elem_op_2::identity);
    }

    hipMemcpy(data_d, data_h, sizeof(data_t)*len_data, H2D); CUERR

    //retrieve results
    TIMERSTART(op2)
    plan_t::table_operation<table_op_2,
                            elem_op_2>
    (data_d, len_data, hash_table_d, capacity, failure_handler, 0, config);
    TIMERSTOP(op2)

    hipMemcpy(data_h, data_d, sizeof(data_t)*len_data, D2H); CUERR

    //validation
    index_t num_errors = 0;
    //#pragma omp parallel for reduction(+:num_errors)
    for (index_t i = 0; i < len_data; i++) {
        if (data_h[i].get_value() != i+1)
        {
            num_errors += 1;
        }
    }

    cout << "ERRORS: " << num_errors << endl;

    //free memory
    delete[] keys_h;
    delete[] data_h;

    hipFree(hash_table_d);
    hipFree(data_d);
}
